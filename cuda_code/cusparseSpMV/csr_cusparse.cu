#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h> 
#include <bits/stdc++.h>
#include "utilities.h"
#include <sys/time.h>
#include "io.h"

using namespace std;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main(int argc, char ** argv) {
    unsigned int n, m, nnz = 0;
    unsigned int block_num = 0;
    unsigned int nnz_max;
    float *X;

    string file_name = argv[1];

    conv(file_name, nnz, m, n, nnz_max, block_num);

    cout << "nnz:" << nnz << endl;
    cout << "row_num:" << m << endl;
    cout << "col_num:" << n << endl;
    cout << "max_row_length:" << nnz_max << endl;
    cout << "block_num:" << block_num << endl;

    X = vect_gen(n);
    float *Y = (float *)malloc(m * sizeof(float));

    for (int i = 0; i < m; i++)
    {
        Y[i] = 0;
    }

    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (m + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  nnz * sizeof(float))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         n * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         m * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, row_off,
                           (m + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, col_idx, nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, values, nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, X, n * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, Y, m * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, m, n, nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, n, dX, HIP_R_32F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, m, dY, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    hipDeviceSynchronize();

	struct timeval start,end;
	gettimeofday(&start, NULL);

    int repeat_num = 5000;

    for (unsigned int i = 0; i < repeat_num; i++)
	{
        // execute SpMV
        CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer) )
        hipDeviceSynchronize();
    }

    gettimeofday(&end, NULL);

    long timeuse = 1000000 * (end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
	double gflops = ((double)2.0 * nnz * repeat_num / ((double)timeuse / 1000000)) / 1000000000;

	float exe_time = (float)timeuse / 1000.0;
	float exe_gflops = gflops;

    printf("time=%fms, gflops=%f\n", exe_time, exe_gflops);

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return 1;
}