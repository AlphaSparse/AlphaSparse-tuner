#include "hip/hip_runtime.h"
#include <iostream>

#include "anonymouslib_cuda.h"
#include <sys/time.h>
#include "mmio.h"

using namespace std;

#ifndef VALUE_TYPE
#define VALUE_TYPE double
#endif

#ifndef NUM_RUN
#define NUM_RUN 1000
#endif

int call_anonymouslib(int m, int n, int nnzA,
                  int *csrRowPtrA, int *csrColIdxA, VALUE_TYPE *csrValA,
                  VALUE_TYPE *x, VALUE_TYPE *y, VALUE_TYPE alpha)
{
    int err = 0;
    hipError_t err_cuda = hipSuccess;

    // set device
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    cout << "Device [" <<  device_id << "] " << deviceProp.name << ", " << " @ " << deviceProp.clockRate * 1e-3f << "MHz. " << endl;

    double gb = getB<int, VALUE_TYPE>(m, nnzA);
    double gflop = getFLOP<int>(nnzA);

    // Define pointers of matrix A, vector x and y
    int *d_csrRowPtrA;
    int *d_csrColIdxA;
    VALUE_TYPE *d_csrValA;
    VALUE_TYPE *d_x;
    VALUE_TYPE *d_y;

    // Matrix A
    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, (m+1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIdxA, nnzA  * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_csrValA,    nnzA  * sizeof(VALUE_TYPE)));

    checkCudaErrors(hipMemcpy(d_csrRowPtrA, csrRowPtrA, (m+1) * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrColIdxA, csrColIdxA, nnzA  * sizeof(int),   hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrValA,    csrValA,    nnzA  * sizeof(VALUE_TYPE),   hipMemcpyHostToDevice));

    // Vector x
    checkCudaErrors(hipMalloc((void **)&d_x, n * 2 * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemcpy(d_x, x, n * 2 * sizeof(VALUE_TYPE), hipMemcpyHostToDevice));

    // Vector y
    checkCudaErrors(hipMalloc((void **)&d_y, m*2  * sizeof(VALUE_TYPE)));
    checkCudaErrors(hipMemset(d_y, 0, m*2 * sizeof(VALUE_TYPE)));

    // 数组A
    anonymouslibHandle<int, unsigned int, VALUE_TYPE> A(m, n);
    err = A.inputCSR(nnzA, d_csrRowPtrA, d_csrColIdxA, d_csrValA);
    //cout << "inputCSR err = " << err << endl;

    err = A.setX(d_x); // you only need to do it once!
    //cout << "setX err = " << err << endl;

    A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);

    // warmup device
    // A.warmup();

    anonymouslib_timer asCSR5_timer;
    asCSR5_timer.start();

    err = A.asCSR5();

    cout << "CSR->CSR5 time = " << asCSR5_timer.stop() << " ms." << endl;
    //cout << "asCSR5 err = " << err << endl;

    // check correctness by running 1 time
    err = A.spmv(alpha, d_y);
    cout << "spmv err = " << err << endl;
    checkCudaErrors(hipMemcpy(y, d_y, m * sizeof(VALUE_TYPE), hipMemcpyDeviceToHost));

    // warm up by running 50 times
    // if (NUM_RUN)
    // {
    //     for (int i = 0; i < 50; i++)
    //         err = A.spmv(alpha, d_y);
    // }

    err_cuda = hipDeviceSynchronize();

    struct timeval start, end;
    gettimeofday(&start, NULL);
    anonymouslib_timer CSR5Spmv_timer;
    CSR5Spmv_timer.start();

    // time spmv by running NUM_RUN times
    for (int i = 0; i < NUM_RUN; i++)
    {
        err = A.spmv(alpha, d_y);
        err_cuda = hipDeviceSynchronize();
    }

    double CSR5Spmv_time = CSR5Spmv_timer.stop() / (double)NUM_RUN;

    gettimeofday(&end, NULL);

    long timeuse = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    double gflops = ((double)2.0 * nnzA * NUM_RUN / ((double)timeuse / 1000000)) / 1000000000;

    // 时间和性能
    float exe_time = (float)timeuse / 1000.0;
    float exe_gflops = gflops;

    // 循环的次数
    int final_repeat_num = NUM_RUN * ((float)1000 / exe_time);

    gettimeofday(&start, NULL);

    for (int i = 0; i < final_repeat_num; i++)
    {
        err = A.spmv(alpha, d_y);
        err_cuda = hipDeviceSynchronize();
    }
    
    gettimeofday(&end, NULL);

    timeuse = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    gflops = ((double)2.0 * nnzA * final_repeat_num / ((double)timeuse / 1000000)) / 1000000000;

    exe_time = (float)timeuse / 1000.0;
    exe_gflops = gflops;

    printf("CSR5:time=%fms, gflops=%f\n", exe_time, exe_gflops);

    A.destroy();

    checkCudaErrors(hipFree(d_csrRowPtrA));
    checkCudaErrors(hipFree(d_csrColIdxA));
    checkCudaErrors(hipFree(d_csrValA));
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));

    return err;
}

int main(int argc, char ** argv)
{
    int m, n, nnzA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;

    // report precision of floating-point
    cout << "------------------------------------------------------" << endl;
    char  *precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = "32-bit Single Precision";
    }
    else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = "64-bit Double Precision";
    }
    else
    {
        cout << "Wrong precision. Program exit!" << endl;
        return 0;
    }

    cout << "PRECISION = " << precision << endl;
    cout << "------------------------------------------------------" << endl;

    //ex: ./spmv webbase-1M.mtx
    int argi = 1;

    char  *filename;
    if(argc > argi)
    {
        filename = argv[argi];
        argi++;
    }
    cout << "--------------" << filename << "--------------" << endl;

    // read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE *f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;

    // load matrix
    if ((f = fopen(filename, "r")) == NULL)
    {
        return -1;
    }

    // if (mm_read_banner(f, &matcode) != 0)
    // {
    //     cout << "Could not process Matrix Market banner." << endl;
    //     return -2;
    // }

    // if ( mm_is_complex( matcode ) )
    // {
    //     cout <<"Sorry, data type 'COMPLEX' is not supported. " << endl;
    //     return -3;
    // }

    // if ( mm_is_pattern( matcode ) )  { isPattern = 1; /*cout << "type = Pattern" << endl;*/ }
    // if ( mm_is_real ( matcode) )     { isReal = 1; /*cout << "type = real" << endl;*/ }
    // if ( mm_is_integer ( matcode ) ) { isInteger = 1; /*cout << "type = integer" << endl;*/ }

    // /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
    {
        return -4;
    }

    nnzA = nnzA_mtx_report;

    // if ( mm_is_symmetric( matcode ) || mm_is_hermitian( matcode ) )
    // {
    //     isSymmetric = 1;
    //     //cout << "symmetric = true" << endl;
    // }
    // else
    // {
    //     //cout << "symmetric = false" << endl;
    // }

    // int *csrRowPtrA_counter = (int *)malloc((m+1) * sizeof(int));
    // memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    // int *csrRowIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    // int *csrColIdxA_tmp = (int *)malloc(nnzA_mtx_report * sizeof(int));
    // VALUE_TYPE *csrValA_tmp    = (VALUE_TYPE *)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    // /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    // /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    // /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    // for (int i = 0; i < nnzA_mtx_report; i++)
    // {
    //     int idxi, idxj;
    //     double fval;
    //     int ival;

    //     if (isReal)
    //         fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
    //     else if (isInteger)
    //     {
    //         fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
    //         fval = ival;
    //     }
    //     else if (isPattern)
    //     {
    //         fscanf(f, "%d %d\n", &idxi, &idxj);
    //         fval = 1.0;
    //     }

    //     // adjust from 1-based to 0-based
    //     idxi--;
    //     idxj--;

    //     csrRowPtrA_counter[idxi]++;
    //     csrRowIdxA_tmp[i] = idxi;
    //     csrColIdxA_tmp[i] = idxj;
    //     csrValA_tmp[i] = fval;
    // }

    // if (f != stdin)
    //     fclose(f);

    // if (isSymmetric)
    // {
    //     for (int i = 0; i < nnzA_mtx_report; i++)
    //     {
    //         if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
    //             csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
    //     }
    // }

    // // exclusive scan for csrRowPtrA_counter
    // int old_val, new_val;

    // old_val = csrRowPtrA_counter[0];
    // csrRowPtrA_counter[0] = 0;
    // for (int i = 1; i <= m; i++)
    // {
    //     new_val = csrRowPtrA_counter[i];
    //     csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i-1];
    //     old_val = new_val;
    // }

    // nnzA = csrRowPtrA_counter[m];
    // csrRowPtrA = (int *)malloc((m+1) * sizeof(int));
    // memcpy(csrRowPtrA, csrRowPtrA_counter, (m+1) * sizeof(int));
    // memset(csrRowPtrA_counter, 0, (m+1) * sizeof(int));

    // csrColIdxA = (int *)malloc(nnzA * sizeof(int));
    // csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));

    // if (isSymmetric)
    // {
    //     for (int i = 0; i < nnzA_mtx_report; i++)
    //     {
    //         if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
    //         {
    //             int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
    //             csrColIdxA[offset] = csrColIdxA_tmp[i];
    //             csrValA[offset] = csrValA_tmp[i];
    //             csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

    //             offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
    //             csrColIdxA[offset] = csrRowIdxA_tmp[i];
    //             csrValA[offset] = csrValA_tmp[i];
    //             csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
    //         }
    //         else
    //         {
    //             int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
    //             csrColIdxA[offset] = csrColIdxA_tmp[i];
    //             csrValA[offset] = csrValA_tmp[i];
    //             csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
    //         }
    //     }
    // }
    // else
    // {
    //     for (int i = 0; i < nnzA_mtx_report; i++)
    //     {
    //         int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
    //         csrColIdxA[offset] = csrColIdxA_tmp[i];
    //         csrValA[offset] = csrValA_tmp[i];
    //         csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
    //     }
    // }

    // // free tmp space
    // free(csrColIdxA_tmp);
    // free(csrValA_tmp);
    // free(csrRowIdxA_tmp);
    // free(csrRowPtrA_counter);

    // srand(time(NULL));

    // 这里直接进行文件的读取，并且获得CSR的三个数组
    


    // set csrValA to 1, easy for checking floating-point results
    // for (int i = 0; i < nnzA; i++)
    // {
    //     csrValA[i] = rand() % 10;
    // }

    cout << " ( " << m << ", " << n << " ) nnz = " << nnzA << endl;


    int max_row_length;
    int *col_idx = NULL;
    int *row_off = NULL;
    
    // 强制使用单精度
    VALUE_TYPE *values = NULL;

    // 申请对应大小的数组
    col_idx = new int[nnzA];
    row_off = new int[m + 1];
    values = new VALUE_TYPE[nnzA];
    
    // 将数据读到三个矩阵中
    conv(filename, row_off, col_idx, values, max_row_length, true);

    cout << " ( " << m << ", " << n << " ) nnz = " << nnzA << ", max row length:" << max_row_length << endl;

    // exit(-1);
    csrRowPtrA = row_off;
    csrColIdxA = col_idx;
    csrValA = values;

    // 打印几个关键的数组
    for (int i; i < 10; i++)
    {
        cout << csrRowPtrA[i] << ",";
    }

    cout << endl;

    for (int i; i < 10; i++)
    {
        cout << csrColIdxA[i] << ",";
    }

    cout << endl;
    
    for (int i; i < 10; i++)
    {
        cout << csrValA[i] << ",";
    }

    cout << endl;

    // exit(-1);

    VALUE_TYPE *x = (VALUE_TYPE *)malloc(n * sizeof(VALUE_TYPE));
    for (int i = 0; i < n; i++)
        x[i] = 1;

    VALUE_TYPE *y = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));
    VALUE_TYPE *y_ref = (VALUE_TYPE *)malloc(m * sizeof(VALUE_TYPE));

    double gb = getB<int, VALUE_TYPE>(m, nnzA);
    double gflop = getFLOP<int>(nnzA);

    VALUE_TYPE alpha = 1.0;

    // compute reference results on a cpu core
    anonymouslib_timer ref_timer;
    ref_timer.start();

    int ref_iter = 1;
    // for (int iter = 0; iter < ref_iter; iter++)
    // {
    //     for (int i = 0; i < m; i++)
    //     {
    //         VALUE_TYPE sum = 0;
    //         for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
    //             sum += x[csrColIdxA[j]] * csrValA[j] * alpha;
    //         y_ref[i] = sum;
    //     }
    // }

    double ref_time = ref_timer.stop() / (double)ref_iter;
    // cout << "cpu sequential time = " << ref_time
    //      << " ms. Bandwidth = " << gb/(1.0e+6 * ref_time)
    //      << " GB/s. GFlops = " << gflop/(1.0e+6 * ref_time)  << " GFlops." << endl << endl;

    // 在这之前都是预处理
    gettimeofday(&pre_end, NULL);

    // 预处理时间计算
    double pre_timeuse = 1000000 * (pre_end.tv_sec - pre_start.tv_sec) + pre_end.tv_usec - pre_start.tv_usec;

    printf("pre_process_time=%fms\n", pre_timeuse / 1000.0);
    // launch compute
    call_anonymouslib(m, n, nnzA, csrRowPtrA, csrColIdxA, csrValA, x, y, alpha);

    // compare reference and anonymouslib results
    if (false)
    {
        int error_count = 0;
        for (int i = 0; i < m; i++)
            if (abs(y_ref[i] - y[i]) > 0.01 * abs(y_ref[i]))
            {
                error_count++;
    //            cout << "ROW [ " << i << " ], NNZ SPAN: "
    //                 << csrRowPtrA[i] << " - "
    //                 << csrRowPtrA[i+1]
    //                 << "\t ref = " << y_ref[i]
    //                 << ", \t csr5 = " << y[i]
    //                 << ", \t error = " << y_ref[i] - y[i]
    //                 << endl;
    //            break;

    //            //if (abs(y_ref[i] - y[i]) > 0.00001)
    //            //    cout << ", \t error = " << y_ref[i] - y[i] << endl;
    //            //else
    //            //    cout << ". \t CORRECT!" << endl;
            }

        if (error_count == 0)
            cout << "Check... PASS!" << endl;
        else
            cout << "Check... NO PASS! #Error = " << error_count << " out of " << m << " entries." << endl;

        cout << "------------------------------------------------------" << endl;
    }

     for (int i; i < 10; i++)
    {
        cout << y[i] << ",";
    }

    

    cout << endl;

    free(csrRowPtrA);
    free(csrColIdxA);
    free(csrValA);
    free(x);
    free(y);
    free(y_ref);

    return 0;
}

